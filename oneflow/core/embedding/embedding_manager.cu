/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/embedding/embedding_manager.h"
#include "oneflow/core/embedding/persistent_table_key_value_store.h"
#include "oneflow/core/ep/include/device_manager_registry.h"
#include "oneflow/core/embedding/cached_key_value_store.h"

namespace oneflow {

namespace embedding {}  // namespace embedding

EmbeddingManager::~EmbeddingManager() {
  for (auto& pair : key_value_store_map_) { pair.second->SaveSnapshot("index"); }
}

embedding::KeyValueStore* EmbeddingManager::GetKeyValueStore(const std::string& embedding_name,
                                                             int64_t parallel_id) {
  OF_CUDA_CHECK(hipSetDevice(parallel_id));
  std::pair<std::string, int64_t> map_key = std::make_pair(embedding_name, parallel_id);
  std::unique_lock<std::mutex> lock(mutex_);
  auto it = key_value_store_map_.find(map_key);
  return it->second.get();
}

void EmbeddingManager::CreateKeyValueStore(
    const embedding::KeyValueStoreOptions& key_value_store_options, int64_t parallel_id,
    int64_t parallel_num) {
  OF_CUDA_CHECK(hipSetDevice(parallel_id));
  const std::string& name = key_value_store_options.Name();
  const uint32_t line_size = key_value_store_options.LineSize();
  std::pair<std::string, int64_t> map_key = std::make_pair(name, parallel_id);
  std::unique_lock<std::mutex> lock(mutex_);

  std::unique_ptr<embedding::KeyValueStore> store;
  const std::string& path = key_value_store_options.PersistentTablePath();
  const std::string& num_rank = std::to_string(parallel_num);
  const int32_t rank_id_suffix_length = num_rank.size();
  const std::string& rank_id = std::to_string(parallel_id);
  embedding::PersistentTableKeyValueStoreOptions options{};
  options.table_options.path = path + "/" + std::string(rank_id_suffix_length - rank_id.size(), '0')
                               + rank_id + "-" + num_rank;
  options.table_options.value_size = line_size * GetSizeOfDataType(DataType::kFloat);
  options.table_options.key_size = GetSizeOfDataType(DataType::kInt64);
  options.table_options.physical_block_size =
      key_value_store_options.PersistentTablePhysicalBlockSize();
  options.table_options.target_chunk_size_mb = 4 * 1024;
  store = NewPersistentTableKeyValueStore(options);
  const std::vector<embedding::CacheOptions> cache_options =
      key_value_store_options.GetCachesOptions();
  for (int i = 0; i < cache_options.size(); ++i) {
    std::unique_ptr<embedding::Cache> cache = embedding::NewCache(cache_options.at(i));
    LOG(ERROR) << "add cache: " << cache_options.at(i).policy << " "
               << cache_options.at(i).capacity;
    store = NewCachedKeyValueStore(std::move(store), std::move(cache));
  }
  key_value_store_map_.emplace(map_key, std::move(store));
}

void EmbeddingManager::SaveSnapshot(const std::string& embedding_name, int64_t parallel_id,
                                    const std::string& snapshot_name) {
  OF_CUDA_CHECK(hipSetDevice(parallel_id));
  std::pair<std::string, int64_t> map_key = std::make_pair(embedding_name, parallel_id);
  std::unique_lock<std::mutex> lock(mutex_);

  auto it = key_value_store_map_.find(map_key);
  if (it != key_value_store_map_.end()) {
    it->second->SaveSnapshot(snapshot_name);
  } else {
    LOG(ERROR) << "Can not find embedding: " << embedding_name << "-" << parallel_id;
  }
}

void EmbeddingManager::LoadSnapshot(const std::string& embedding_name, int64_t parallel_id,
                                    const std::string& snapshot_name) {
  OF_CUDA_CHECK(hipSetDevice(parallel_id));
  std::pair<std::string, int64_t> map_key = std::make_pair(embedding_name, parallel_id);
  auto it = key_value_store_map_.find(map_key);
  if (it != key_value_store_map_.end()) {
    if (it->second->SnapshotExists(snapshot_name)) {
      it->second->LoadSnapshot(snapshot_name);
    } else {
      LOG(ERROR) << "Here Exists Embedding name is: " << embedding_name << "-" << parallel_id
                 << " but no corresponding snapshot. ";
    }
  } else {
    LOG(ERROR) << "Can not find the embedding: " << embedding_name << "-" << parallel_id;
  }
}

}  // namespace oneflow
